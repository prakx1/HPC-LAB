#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#define row1 500 
#define col1 500  
#define row2 500 
#define col2 500 

__global__ void matrix_multiply(int *l,int *m, int *n)
{
    int x=blockIdx.x;
    int y=blockIdx.y;
    int k;
  
n[col2*y+x]=0;
for(k=0;k<col1;k++)
   {
    n[col2*y+x]=n[col2*y+x]+l[col1*y+k]*m[col2*k+x];
   }
}

int main()
{
    int a[row1][col1];
    int b[row2][col2];
    int c[row1][col2];
    int *d,*e,*f;
    int i,j;

    for(i=0;i<row1;i++)
    {
        for(j=0;j<col1;j++)
            {
                a[i][j]=1;
            }
    }
    
        for(i=0;i<row2;i++)
        {
            for(j=0;j<col2;j++)
                {
                    b[i][j]=1;
                }
        }

    double time_spent = 0.0;
    clock_t begin = clock();
 
    hipMalloc((void **)&d,row1*col1*sizeof(int));
    hipMalloc((void **)&e,row2*col2*sizeof(int));
    hipMalloc((void **)&f,row1*col2*sizeof(int));

 hipMemcpy(d,a,row1*col1*sizeof(int),hipMemcpyHostToDevice);
 hipMemcpy(e,b,row2*col2*sizeof(int),hipMemcpyHostToDevice);

dim3 grid(col2,row1);
    matrix_multiply<<<grid,1>>>(d,e,f);

 hipMemcpy(c,f,row1*col2*sizeof(int),hipMemcpyDeviceToHost);
 clock_t end = clock();

    hipFree(d);
    hipFree(e);
    hipFree(f);
 
    time_spent += (double)(end - begin) / CLOCKS_PER_SEC;
 
    printf("The elapsed time is %f seconds\n", time_spent);

    return 0;
}

