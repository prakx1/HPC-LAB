#include<iostream>
#include<hip/hip_runtime.h>
using namespace std;

__global__ void staticmultiply(int *x, int n)
{
  __shared__ int tmp[64];
  int index = threadIdx.x;
  tmp[index] = x[index]*100;
  __syncthreads();
  x[index] = tmp[index];
}

__global__ void dynamicmultiply(int *x, int n)
{
  extern __shared__ int tmp[];
  int index = threadIdx.x;
  tmp[index] = x[index]*100;
  __syncthreads();
  x[index] = tmp[index];
}

int main(void)
{
  const int n=10;
  cout<<"Elements in array:";
  
  int a[n],x[n];

  for (int i = 0; i < n; i++) {
    a[i]=i;
  }

  int *arr;
  hipMalloc(&arr, n * sizeof(int)); 

 
  hipMemcpy(arr, a, n*sizeof(int), hipMemcpyHostToDevice);
  staticmultiply<<<1,n>>>(arr, n);
  hipMemcpy(x, arr, n*sizeof(int), hipMemcpyDeviceToHost);
  for (int i = 0; i < n; i++) 
    {
      cout<<x[i]<<" ";
    }
  cout<<"\n";
 
  hipMemcpy(arr, a, n*sizeof(int), hipMemcpyHostToDevice);
  dynamicmultiply<<<1,n,n*sizeof(int)>>>(arr, n);
  hipMemcpy(x, arr, n * sizeof(int), hipMemcpyDeviceToHost);
  for (int i = 0; i < n; i++) 
    {
      cout<<x[i]<<" ";
    }
  cout<<"\n";
  return 0;
}
